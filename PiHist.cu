/**
 * This program implements a parallel code via a kernel function call that computes the distribution of the digits of Pi
 * using hierarchical atomic strategy.
 *
 * Users are expected to enter three arguments: the executable file, the filename that contains 10 million digits of pi,
 * and the number of digits to be evaluated.
 *
 * @author Richard Zhang {zhank20@wfu.edu}
 * @date Mar.15, 2023
 * @assignment Lab 3
 * @course CSC 347
 **/

#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024

__global__ void computeFrequency(int* de_frequency, char* de_digits, int numDigits) {
    // Initialize shared memory for de_frequency
    __shared__ int local[10];

    for (int i = 0; i < 10; i ++) {
        local[i] = 0;
    }
    __syncthreads();

    // Update shared de_frequency
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate the digits we need to skip because one thread only need to compute a part of all the digits that need to be processed

    char digit = de_digits[index];
    if (index < numDigits && digit >= '0' && digit <= '9') {
        local[digit-'0']++;
    }
    __syncthreads();

    // Update global de_frequency with shared local array
    for (int i = 0; i < 10; i++) {
        atomicAdd(&(de_frequency[i]), local[i]);
        printf("test: %d", local[i]);
    }
    printf("fuku %d", 1);
}

int main(int argc, char *argv[]) {
    // Determine if there are two arguments on the command line
    if (argc != 3) {
        printf("Command line arguments are not enough: %s \n", argv[0]);
        return 1;
    }

    // Convert the third argument to integer
    int numDigits = atoi(argv[2]);

    // Determine if the number of digits entered by users is legitimate
    if (numDigits <= 0) {
        printf("Number of digits should not be less than 1\n");
        return 2;
    }

    // Check if we can open the file. If not, return error message
    FILE *fp;
    fp = fopen(argv[1], "r");
    if (fp == NULL) {
        printf("%s could not be opened\n", argv[1]);
        exit(1);
    }

    // Allocate memory for digit buffer and read in the digits
    char *digits = (char *) malloc(numDigits * sizeof(char));
    // Pass all the number read to the array digits
    fread(digits, sizeof(char), numDigits, fp);

    // Allocate memory for de_frequency
    int *de_frequency;
    int *frequency;
    frequency=(int*)malloc(10*sizeof(int));
    hipMalloc((void**)&de_frequency, 10 * sizeof(int));
    hipMemset(de_frequency, 0, 10 * sizeof(int));

    // Compute the distribution of digits using CUDA kernel
    int gridSize = (numDigits + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int blockSize = BLOCK_SIZE;
    computeFrequency<<<gridSize, blockSize>>>(de_frequency, digits, numDigits);
    hipDeviceSynchronize();

    hipDeviceSynchronize();
    hipMemcpy(frequency, de_frequency, 10*sizeof(int), hipMemcpyDeviceToHost);

    // Print the resulting frequency of digits
    for (int i = 0; i < 10; i++) {
        // Print the digit frequency in the format of digit: count
       printf("%d:\t%d\n", i, frequency[i]);;
    }

    // Clean up
    free(digits);
    free(frequency);
    hipFree(de_frequency);
    fclose(fp);

    return 0;
}
